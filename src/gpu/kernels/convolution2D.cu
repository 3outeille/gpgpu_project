#include "hip/hip_runtime.h"
#include "../kernels.cuh"
#include "helpers.cuh"

template <int KERNEL_SIZE>
__global__ void convolution_2D_gpu_kernel(float *output_buffer, const float *input, int width, int height, size_t pitch, const float *kernel)
{
	int i = threadIdx.y + blockIdx.y * blockDim.y;
	int j = threadIdx.x + blockIdx.x * blockDim.x;

	if (i >= height || j >= width)
		return;

	float cell_value = 0;

	int size = KERNEL_SIZE / 2;

	for (int k_i = -size; k_i <= size && i + k_i < height; ++k_i)
	{
		if (i + k_i < 0)
			continue;
		for (int k_j = -size; k_j <= size && j + k_j < width; ++k_j)
		{
			if (j + k_j < 0)
				continue;
			float image_value = *eltPtr(input, j + k_j, i + k_i, pitch);
			auto kernel_value = kernel[(k_i + size) * KERNEL_SIZE + (k_j + size)];
			cell_value += image_value * kernel_value;
		}
	}
	output_buffer[i * width + j] = cell_value;
}

template <int KERNEL_SIZE>
__global__ void convolution_2D_tiled_gpu_kernel(float *output_buffer, const float *input, int width, int height, size_t pitch, const float *kernel)
{
	extern __shared__ float kernel_cached[];
	__shared__ float padded_tile[BLOCK_SIZE][BLOCK_SIZE];

	if (threadIdx.y < KERNEL_SIZE && threadIdx.x < KERNEL_SIZE)
		kernel_cached[threadIdx.y * KERNEL_SIZE + threadIdx.x] = kernel[threadIdx.y * KERNEL_SIZE + threadIdx.x];

	int tile_size = BLOCK_SIZE - KERNEL_SIZE + 1;
	int input_i = threadIdx.y + blockIdx.y * tile_size;
	int input_j = threadIdx.x + blockIdx.x * tile_size;

	// Load tile
	int block_i = input_i - (KERNEL_SIZE / 2);
	int block_j = input_j - (KERNEL_SIZE / 2);

	if (block_i >= 0 && block_i < height && block_j >= 0 && block_j < width)
		padded_tile[threadIdx.y][threadIdx.x] = *eltPtr(input, block_j, block_i, pitch);
	else
		padded_tile[threadIdx.y][threadIdx.x] = 0.0f;

	__syncthreads();

	if (threadIdx.x >= tile_size || threadIdx.y >= tile_size || input_i >= height || input_j >= width)
		return;

	float cell_value = 0.;

	for (int k_i = 0; k_i < KERNEL_SIZE; ++k_i)
		for (int k_j = 0; k_j < KERNEL_SIZE; ++k_j)
			cell_value += padded_tile[threadIdx.y + k_i][threadIdx.x + k_j] * kernel_cached[k_i * KERNEL_SIZE + k_j];

	output_buffer[input_i * width + input_j] = cell_value;
}

template<int KERNEL_SIZE>
__global__ void convolution_2D_tiled_loop_gpu_kernel(float *output_buffer, const float *input, int width, int height, size_t pitch, const float *kernel)
{
	const int PADDED_TILE_SIZE = BLOCK_SIZE + KERNEL_SIZE - 1;
	const int HALF_KERNEL = KERNEL_SIZE / 2;

	__shared__ float padded_tile[PADDED_TILE_SIZE][PADDED_TILE_SIZE];
	__shared__ float kernel_cached[KERNEL_SIZE][KERNEL_SIZE];

	for (int i = threadIdx.y; i < PADDED_TILE_SIZE; i += BLOCK_SIZE)
	{
		for (int j = threadIdx.x; j < PADDED_TILE_SIZE; j += BLOCK_SIZE)
		{
			int input_i = threadIdx.y + (blockIdx.y * BLOCK_SIZE) - HALF_KERNEL;
			int input_j = threadIdx.x + (blockIdx.x * BLOCK_SIZE) - HALF_KERNEL;

			if (input_i < 0 || input_i >= height || input_j < 0 || input_j >= width)
				padded_tile[i][j] = 0;
			else
				padded_tile[i][j] = *eltPtr(input, input_j, input_i, pitch);
		}
	}

	if (threadIdx.y < KERNEL_SIZE && threadIdx.x < KERNEL_SIZE)
		kernel_cached[threadIdx.y][threadIdx.x] = kernel[threadIdx.y * KERNEL_SIZE + threadIdx.x];

	__syncthreads();

	auto output_i = threadIdx.y + blockIdx.y * BLOCK_SIZE;
	auto output_j = threadIdx.x + blockIdx.x * BLOCK_SIZE;

	if (output_i >= height || output_j >= width)
		return;

	float cell_value = 0.;

	for (int k_i = 0; k_i < KERNEL_SIZE; ++k_i)
		for (int k_j = 0; k_j < KERNEL_SIZE; ++k_j)
			cell_value += padded_tile[threadIdx.y + k_i][threadIdx.x + k_j] * kernel_cached[k_i][k_j];

	output_buffer[output_i * width + output_j] = cell_value;
}

MatrixGPU convolution_2D_gpu(MatrixGPU &input, MatrixGPU &kernel)
{
	float *input_pitched;
	size_t pitch;
	hipMallocPitch(&input_pitched, &pitch, input.width * sizeof(float), input.height);
	hipMemcpy2D(input_pitched, pitch, input.to_device_buffer(), input.width * sizeof(float), input.width * sizeof(float), input.height, hipMemcpyDeviceToDevice);

	MatrixGPU output(input.height, input.width);

	convolution_2D_gpu_kernel<KERNEL_SIZE><<<output.dimGrid(), output.dimBlock()>>>(output.to_device_buffer(), input_pitched, input.width, input.height, pitch, kernel.to_device_buffer());


	// MatrixGPU output(input.height, input.width);

	// auto dim_block = dim3(BLOCK_SIZE, BLOCK_SIZE);
	// float tile_size = BLOCK_SIZE - kernel.width + 1;
	// auto dim_grid_width = std::ceil((float)input.width / tile_size);
	// auto dim_grid_height = std::ceil((float)input.height / tile_size);
	// auto dim_grid = dim3(dim_grid_width, dim_grid_height);
	// convolution_2D_tiled_gpu_kernel<KERNEL_SIZE><<<dim_grid, dim_block, kernel.width * kernel.width>>>(output.to_device_buffer(), input_pitched, input.width, input.height, pitch, kernel.to_device_buffer());

	// MatrixGPU output(input.height, input.width);
	// convolution_2D_tiled_loop_gpu_kernel<KERNEL_SIZE><<<input.dimGrid(), input.dimBlock()>>>(output.to_device_buffer(), input_pitched, input.width, input.height, pitch, kernel.to_device_buffer());

	// hipDeviceSynchronize();

	if (hipPeekAtLastError())
		abortError("Computation Error");

	hipFree(input_pitched);

	return output;
}
