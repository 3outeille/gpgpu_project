#include "hip/hip_runtime.h"
#include "render.hpp"
#include "matrix.cuh"
#include <spdlog/spdlog.h>
#include <cassert>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <cstdio>

[[gnu::noinline]] void _abortError(const char *msg, const char *fname, int line)
{
  hipError_t err = hipGetLastError();
  spdlog::error("{} ({}, line: {})", msg, fname, line);
  spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
  std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)

__global__ void grayscale_gpu_kernel(unsigned char *input_buffer, int width, int height, float *output_buffer)
{
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= height || j >= width)
    return;

  auto index = i * width + j;

  float r = input_buffer[index * 4 + 0];
  float g = input_buffer[index * 4 + 1];
  float b = input_buffer[index * 4 + 2];

  output_buffer[index] = 0.2126 * r + 0.7152 * g + 0.0722 * b;
}

MatrixGPU grayscale_gpu(thrust::device_vector<unsigned char> &input, int width, int height)
{
  unsigned char *input_buffer_raw = thrust::raw_pointer_cast(input.data());

  MatrixGPU output(height, width);
  float *output_buffer_raw = thrust::raw_pointer_cast(output.data.data());

  grayscale_gpu_kernel<<<output.dimGrid(), output.dimBlock()>>>(input_buffer_raw, width, height, output_buffer_raw);
  hipDeviceSynchronize();

  if (hipPeekAtLastError())
    abortError("Computation Error");

  return output;
}

__global__ void gauss_filter_gpu_kernel(float *output_buffer, int kernel_size, int size)
{
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= kernel_size || j >= kernel_size)
    return;

  auto x = j - size;
  auto y = i - size;
  auto left_x = pow(x, 2) / (2. * pow((1. / 3.) * size, 2));
  auto right_y = pow(y, 2) / (2. * pow((1. / 3.) * size, 2));
  output_buffer[i * kernel_size + j] = exp(-(left_x + right_y));
}

MatrixGPU gauss_filter_gpu(int size)
{
  int kernel_size = 2 * size + 1;

  MatrixGPU output(kernel_size, kernel_size);

  float *output_buffer_raw = thrust::raw_pointer_cast(output.data.data());

  gauss_filter_gpu_kernel<<<output.dimGrid(), output.dimBlock()>>>(output_buffer_raw, kernel_size, size);
  hipDeviceSynchronize();

  if (hipPeekAtLastError())
    abortError("Computation Error");

  return output;
}

__global__ void compute_gradient_gpu_kernel(float *output_buffer, const float *kernel, int kernel_size, int size, int axis)
{
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= kernel_size || j >= kernel_size)
    return;

  auto previous_index = (axis == 0) ? ((i - 1) * kernel_size + j) : (i * kernel_size + j - 1);
  auto current_index = (i * kernel_size + j);
  auto next_index = (axis == 0) ? ((i + 1) * kernel_size + j) : (i * kernel_size + j + 1);

  auto local_index = (axis == 0 ? i : j);

  if (local_index == 0)
  {
    output_buffer[current_index] = (kernel[next_index] - kernel[current_index]);
  }
  else if (local_index == kernel_size - 1)
  {
    output_buffer[current_index] = (kernel[current_index] - kernel[previous_index]);
  }
  else
  {
    output_buffer[current_index] = (kernel[next_index] - kernel[previous_index]) / 2.;
  }
}

MatrixGPU compute_gradient_gpu(MatrixGPU input, int size, int axis)
{
  int kernel_size = 2 * size + 1;

  MatrixGPU output(kernel_size, kernel_size);

  float *output_buffer_raw = thrust::raw_pointer_cast(output.data.data());

  compute_gradient_gpu_kernel<<<output.dimGrid(), output.dimBlock()>>>(output_buffer_raw, input.to_device_buffer(), kernel_size, size, axis);
  hipDeviceSynchronize();

  if (hipPeekAtLastError())
    abortError("Computation Error");

  return output;
}

__global__ void convolution_2D_gpu_kernel(float *output_buffer, const float *input, int width, int height, const float *kernel, int kernel_size, int size)
{
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= height || j >= width)
    return;

  float cell_value = 0;

  for (int k_i = -size; k_i <= size && i + k_i < height; ++k_i)
  {
    if (i + k_i < 0)
      continue;
    for (int k_j = -size; k_j <= size && j + k_j < width; ++k_j)
    {
      if (j + k_j < 0)
        continue;
      float image_value = input[(i + k_i) * width + (j + k_j)];
      auto kernel_value = kernel[(k_i + size) * kernel_size + (k_j + size)];
      cell_value += image_value * kernel_value;
    }
  }
  output_buffer[i * width + j] = cell_value;
}

MatrixGPU convolution_2D_gpu(MatrixGPU input, MatrixGPU kernel)
{
  int size = (kernel.width - 1) / 2;
  MatrixGPU output(input.height, input.width);

  float *output_buffer_raw = thrust::raw_pointer_cast(output.data.data());

  convolution_2D_gpu_kernel<<<output.dimGrid(), output.dimBlock()>>>(output_buffer_raw, input.to_device_buffer(), input.width, input.height, kernel.to_device_buffer(), kernel.width, size);
  hipDeviceSynchronize();

  if (hipPeekAtLastError())
    abortError("Computation Error");

  return output;
}

MatrixGPU gauss_derivative_gpu(const MatrixGPU &image, const int &size, const int &axis)
{
  auto gradient = compute_gradient_gpu(gauss_filter_gpu(size), size, axis);
  return convolution_2D_gpu(image, gradient);
}

MatrixGPU compute_harris_response_gpu(const MatrixGPU &image)
{
  int size = 3;
  auto img_x = gauss_derivative_gpu(image, size, 1);
  auto img_y = gauss_derivative_gpu(image, size, 0);

  auto gauss = gauss_filter_gpu(size);

  auto W_xx = convolution_2D_gpu(img_x * img_x, gauss);
  auto W_xy = convolution_2D_gpu(img_x * img_y, gauss);
  auto W_yy = convolution_2D_gpu(img_y * img_y, gauss);

  auto W_det = (W_xx * W_yy) - (W_xy * W_xy);
  auto W_trace = W_xx + W_yy;

  return W_det / (W_trace + 1.);
}

__global__ void circle_filter_gpu_kernel(char *output_buffer, int size)
{
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= size || j >= size)
    return;

  auto y = static_cast<float>(i) + 0.5;
  auto x = static_cast<float>(j) + 0.5;
  auto radius = static_cast<float>(size) / 2;
  auto distance = sqrt(pow(x - radius, 2) + pow(y - radius, 2));
  output_buffer[i * size + j] = distance < radius;
}

std::tuple<thrust::device_vector<char>, int> circle_filter_gpu(int size)
{
  thrust::device_vector<char> output(size * size);

  char *output_buffer_raw = reinterpret_cast<char *>(thrust::raw_pointer_cast(output.data()));

  auto dimGrid = dim3(32, 32);
  int w = std::ceil((float)size / 32);
  auto dimBlock = dim3(w, w);

  circle_filter_gpu_kernel<<<dimGrid, dimBlock>>>(output_buffer_raw, size);
  hipDeviceSynchronize();

  if (hipPeekAtLastError())
    abortError("Computation Error");

  return {output, size};
}

__global__ void morph_apply_gpu_kernel(float *output_buffer, const float *input, int width, int height, const char *kernel, int kernel_size, int mode)
{
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= height || j >= width)
    return;

  auto kernel_center = kernel_size / 2 - 1;

  float value = mode == 0 ? 1.0 : 0.0;
  for (int k_i = 0; k_i < kernel_size; k_i++)
  {
    auto img_i = i + k_i - kernel_center;
    for (int k_j = 0; k_j < kernel_size; k_j++)
    {
      auto img_j = j + k_j - kernel_center;

      if (kernel[k_i * kernel_size + k_j] == 0.)
        continue;

      float img_value = 0;
      if (img_i >= 0 && img_i < height && img_j >= 0 && img_j < width)
        img_value = input[img_i * width + img_j];

      if (mode == 0)
        value = fmin(value, img_value);
      else
        value = fmax(value, img_value);
    }
  }
  output_buffer[i * width + j] = value;
}

MatrixGPU morph_apply_gpu(MatrixGPU input, const std::tuple<thrust::device_vector<char>, int> &kernel, int mode)
{
  // mode => erode: 0, dilate: 1
  MatrixGPU output(input.height, input.width);

  auto kernel_buffer = thrust::raw_pointer_cast(std::get<0>(kernel).data());

  morph_apply_gpu_kernel<<<output.dimGrid(), output.dimBlock()>>>(output.to_device_buffer(), input.to_device_buffer(), input.width, input.height, kernel_buffer, std::get<1>(kernel), mode);
  hipDeviceSynchronize();

  if (hipPeekAtLastError())
    abortError("Computation Error");

  return output;
}

std::unique_ptr<unsigned char[]> render_harris_gpu(unsigned char *input_buffer, int width, int height, std::ptrdiff_t stride, int n_iterations)
{
  thrust::host_vector<unsigned char> input_host(input_buffer, input_buffer + (height * width * 4));
  thrust::device_vector<unsigned char> input_device = input_host;

  spdlog::debug("Compute grayscale gpu ...");
  auto img_grayscale = grayscale_gpu(input_device, width, height);

  spdlog::debug("Compute Harris response gpu ...");
  auto harris_res = compute_harris_response_gpu(img_grayscale);

  auto image_mask = img_grayscale > 0;

  spdlog::debug("Erode shape gpu ...");
  auto min_distance = 25;
  auto eroded_mask = morph_apply_gpu(image_mask, circle_filter_gpu(min_distance * 2), 0);
  auto thresholded_mask = eroded_mask * (harris_res > (0.5 * harris_res.max()));

  spdlog::debug("Dilate Harris response...");
  auto dil = morph_apply_gpu(harris_res, circle_filter_gpu(min_distance), 1);
  auto detect_mask = thresholded_mask * (harris_res == dil);

  auto res = detect_mask * 255;
  return res.to_host_buffer();
}
