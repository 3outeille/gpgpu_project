#include "hip/hip_runtime.h"
#include "render.hpp"
#include "matrix.cuh"
#include <spdlog/spdlog.h>
#include <cassert>
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/execution_policy.h>
#include <cstdio>

[[gnu::noinline]] void _abortError(const char *msg, const char *fname, int line)
{
  hipError_t err = hipGetLastError();
  spdlog::error("{} ({}, line: {})", msg, fname, line);
  spdlog::error("Error {}: {}", hipGetErrorName(err), hipGetErrorString(err));
  std::exit(1);
}

#define abortError(msg) _abortError(msg, __FUNCTION__, __LINE__)

__global__ void grayscale_gpu_kernel(unsigned char *input_buffer, int width, int height, double *output_buffer)
{
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= height || j >= width)
    return;

  auto index = i * width + j;

  double r = input_buffer[index * 4 + 0];
  double g = input_buffer[index * 4 + 1];
  double b = input_buffer[index * 4 + 2];

  output_buffer[index] = 0.2126 * r + 0.7152 * g + 0.0722 * b;
}

MatrixGPU grayscale_gpu(thrust::device_vector<unsigned char> &input, int width, int height)
{
  unsigned char *input_buffer_raw = thrust::raw_pointer_cast(input.data());

  MatrixGPU output(height, width);
  double *output_buffer_raw = thrust::raw_pointer_cast(output.data.data());

  grayscale_gpu_kernel<<<output.dimGrid(), output.dimBlock()>>>(input_buffer_raw, width, height, output_buffer_raw);
  hipDeviceSynchronize();

  if (hipPeekAtLastError())
    abortError("Computation Error");

  return output;
}

__global__ void gauss_filter_gpu_kernel(double *output_buffer, int kernel_size, int size)
{
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= kernel_size || j >= kernel_size)
    return;

  auto x = j - size;
  auto y = i - size;
  auto left_x = pow(x, 2) / (2. * pow((1. / 3.) * size, 2));
  auto right_y = pow(y, 2) / (2. * pow((1. / 3.) * size, 2));
  output_buffer[i * kernel_size + j] = exp(-(left_x + right_y));
}

MatrixGPU gauss_filter_gpu(int size)
{
  int kernel_size = 2 * size + 1;

  MatrixGPU output(kernel_size, kernel_size);

  double *output_buffer_raw = thrust::raw_pointer_cast(output.data.data());

  gauss_filter_gpu_kernel<<<output.dimGrid(), output.dimBlock()>>>(output_buffer_raw, kernel_size, size);
  hipDeviceSynchronize();

  if (hipPeekAtLastError())
    abortError("Computation Error");

  return output;
}

__global__ void compute_gradient_gpu_kernel(double *output_buffer, const double *kernel, int kernel_size, int size, int axis)
{
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= kernel_size || j >= kernel_size)
    return;

  auto previous_index = (axis == 0) ? ((i - 1) * kernel_size + j) : (i * kernel_size + j - 1);
  auto current_index = (i * kernel_size + j);
  auto next_index = (axis == 0) ? ((i + 1) * kernel_size + j) : (i * kernel_size + j + 1);

  auto local_index = (axis == 0 ? i : j);

  if (local_index == 0)
  {
    output_buffer[current_index] = (kernel[next_index] - kernel[current_index]);
  }
  else if (local_index == kernel_size - 1)
  {
    output_buffer[current_index] = (kernel[current_index] - kernel[previous_index]);
  }
  else
  {
    output_buffer[current_index] = (kernel[next_index] - kernel[previous_index]) / 2.;
  }
}

MatrixGPU compute_gradient_gpu(MatrixGPU input, int size, int axis)
{
  int kernel_size = 2 * size + 1;

  MatrixGPU output(kernel_size, kernel_size);

  double *output_buffer_raw = thrust::raw_pointer_cast(output.data.data());

  compute_gradient_gpu_kernel<<<output.dimGrid(), output.dimBlock()>>>(output_buffer_raw, input.to_device_buffer(), kernel_size, size, axis);
  hipDeviceSynchronize();

  if (hipPeekAtLastError())
    abortError("Computation Error");

  return output;
}

__global__ void convolution_2D_gpu_kernel(double *output_buffer, const double *input, int width, int height, const double *kernel, int kernel_size, int size)
{
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= height || j >= width)
    return;

  double cell_value = 0;

  for (int k_i = -size; k_i <= size && i + k_i < height; ++k_i)
  {
    if (i + k_i < 0)
      continue;
    for (int k_j = -size; k_j <= size && j + k_j < width; ++k_j)
    {
      if (j + k_j < 0)
        continue;
      double image_value = input[(i + k_i) * width + (j + k_j)];
      auto kernel_value = kernel[(k_i + size) * kernel_size + (k_j + size)];
      cell_value += image_value * kernel_value;
    }
  }
  output_buffer[i * width + j] = cell_value;
}

MatrixGPU convolution_2D_gpu(MatrixGPU input, MatrixGPU kernel)
{
  int size = (kernel.width - 1) / 2;
  MatrixGPU output(input.height, input.width);

  double *output_buffer_raw = thrust::raw_pointer_cast(output.data.data());

  convolution_2D_gpu_kernel<<<output.dimGrid(), output.dimBlock()>>>(output_buffer_raw, input.to_device_buffer(), input.width, input.height, kernel.to_device_buffer(), kernel.width, size);
  hipDeviceSynchronize();

  if (hipPeekAtLastError())
    abortError("Computation Error");

  return output;
}

MatrixGPU gauss_derivative_gpu(const MatrixGPU &image, const int &size, const int &axis)
{
  auto gradient = compute_gradient_gpu(gauss_filter_gpu(size), size, axis);
  return convolution_2D_gpu(image, gradient);
}

MatrixGPU compute_harris_response_gpu(const MatrixGPU &image)
{
  int size = 3;
  auto img_x = gauss_derivative_gpu(image, size, 1);
  auto img_y = gauss_derivative_gpu(image, size, 0);

  auto gauss = gauss_filter_gpu(size);

  auto W_xx = convolution_2D_gpu(img_x * img_x, gauss);
  auto W_xy = convolution_2D_gpu(img_x * img_y, gauss);
  auto W_yy = convolution_2D_gpu(img_y * img_y, gauss);

  auto W_det = (W_xx * W_yy) - (W_xy * W_xy);
  auto W_trace = W_xx + W_yy;

  return W_det / (W_trace + 1.);
}

__global__ void circle_filter_gpu_kernel(double *output_buffer, int size)
{
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= size || j >= size)
    return;

  auto y = static_cast<double>(i) + 0.5;
  auto x = static_cast<double>(j) + 0.5;
  auto radius = static_cast<double>(size) / 2;
  auto distance = sqrt(pow(x - radius, 2) + pow(y - radius, 2));
  output_buffer[i * size + j] = distance < radius;
}

MatrixGPU circle_filter_gpu(int size)
{
  MatrixGPU output(size, size);

  double *output_buffer_raw = thrust::raw_pointer_cast(output.data.data());

  circle_filter_gpu_kernel<<<output.dimGrid(), output.dimBlock()>>>(output_buffer_raw, size);
  hipDeviceSynchronize();

  if (hipPeekAtLastError())
    abortError("Computation Error");

  return output;
}

__global__ void morph_apply_gpu_kernel(double *output_buffer, const double *input, int width, int height, const double *kernel, int kernel_size, int size, int mode)
{
  int i = threadIdx.y + blockIdx.y * blockDim.y;
  int j = threadIdx.x + blockIdx.x * blockDim.x;

  if (i >= height || j >= width)
    return;

  auto half_kernel = kernel_size / 2;

  double value = mode == 0 ? 1.0 : 0.0;
  for (int k_i = 0; k_i < kernel_size; k_i++)
  {
    for (int k_j = 0; k_j < kernel_size; k_j++)
    {
      if (kernel[k_i * kernel_size + k_j] == 0.)
        continue;

      auto img_i = i + k_i - half_kernel;
      auto img_j = j + k_j - half_kernel;
      double img_value = 0;

      if (!(img_i < 0 || img_i >= height || img_j < 0 || img_j >= width))
      {
        img_value = input[img_i * width + img_j];
      }

      value = mode == 0 ? fmin(value, img_value) : fmax(value, img_value);
    }
  }
  output_buffer[i * width + j] = value;
}

MatrixGPU morph_apply_gpu(MatrixGPU input, MatrixGPU kernel, int mode)
{
  // mode => erode: 0, dilate: 1
  int size = (kernel.width - 1) / 2;
  MatrixGPU output(input.height, input.width);

  double *output_buffer_raw = thrust::raw_pointer_cast(output.data.data());

  morph_apply_gpu_kernel<<<output.dimGrid(), output.dimBlock()>>>(output_buffer_raw, input.to_device_buffer(), input.width, input.height, kernel.to_device_buffer(), kernel.width, size, mode);
  hipDeviceSynchronize();

  if (hipPeekAtLastError())
    abortError("Computation Error");

  return output;
}

std::unique_ptr<unsigned char[]> render_harris_gpu(unsigned char *input_buffer, int width, int height, std::ptrdiff_t stride, int n_iterations)
{
  thrust::host_vector<unsigned char> input_host(input_buffer, input_buffer + (height * width * 4));
  thrust::device_vector<unsigned char> input_device = input_host;

  spdlog::debug("Compute grayscale gpu ...");
  auto img_grayscale = grayscale_gpu(input_device, width, height);

  spdlog::debug("Compute Harris response gpu ...");
  auto harris_res = compute_harris_response_gpu(img_grayscale);

  auto image_mask = img_grayscale > 0;

  spdlog::debug("Erode shape gpu ...");
  auto min_distance = 25;
  auto eroded_mask = morph_apply_gpu(image_mask, circle_filter_gpu(min_distance * 2), 0);
  auto thresholded_mask = eroded_mask * (harris_res > (0.5 * harris_res.max()));

  spdlog::debug("Dilate Harris response...");
  auto dil = morph_apply_gpu(harris_res, circle_filter_gpu(min_distance), 1);
  auto detect_mask = thresholded_mask * (harris_res == dil);

  auto res = detect_mask * 255;
  return res.to_host_buffer();
}
